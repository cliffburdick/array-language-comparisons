#include "hip/hip_runtime.h"

#include <matx.h>

auto max_wealth(matx::tensor_t<int32_t, 2> accounts) -> matx::tensor_t<int32_t, 0> {
  auto sums = matx::make_tensor<int32_t>({accounts.Shape()[0]});
  matx::sum(sums, accounts);
  auto max = matx::make_tensor<int32_t>();
  matx::rmax(max, sums);
  return max;
}

auto main() -> int {

  auto t = matx::make_tensor<int32_t>({2, 3});
  t.SetVals({{1, 2, 3}, {3, 2, 1}});
  max_wealth(t).Print(); // 6

  auto t2 = matx::make_tensor<int32_t>({3, 2});
  t2.SetVals({{1, 5}, {7, 3}, {3, 5}});
  max_wealth(t2).Print(); // 10

  auto t3 = matx::make_tensor<int32_t>({3, 3});
  t3.SetVals({{2, 8, 7}, {7, 1, 3}, {1, 9, 5}});
  max_wealth(t3).Print(); // 17

  return 0;
}
